#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime_api.h>
#include"Buffer.h"

template<typename T>inline T min(T a, T b) {
	return a < b ? a : b;
}

inline void*mallocuda(size_t const size) {
	void*result = 0;
	if (hipMalloc(&result, size) != hipSuccess)result = 0;
	return result;
}

template<typename T>Buffer<T>::Buffer(size_t const reserve)throw(int) :length(reserve), devptr((T*)mallocuda(sizeof(T)*reserve)) {
	if (devptr)hipMemset(devptr, 0, sizeof(T)*length);
	else {
		using namespace std;
		cerr << "damepo" << endl; exit(-1);
	}
}

template<typename T>Buffer<T>::Buffer(const Buffer<T>&ref) : Buffer(ref.length) {
	if (devptr)hipMemcpy(devptr, ref.devptr, sizeof(T)*length, hipMemcpyDeviceToDevice);
}

template<typename T>Buffer<T>::Buffer(std::vector<T>&vector) : Buffer(vector.size()) {
	if (devptr)hipMemcpy(devptr, vector.data(), sizeof(T)*length, hipMemcpyHostToDevice);
}

template<typename T>Buffer<T>::~Buffer() {
	if (devptr)hipFree(devptr);
}

template<typename T>Buffer<T>::operator std::vector<T>()const {
	std::vector<T>result(length);
	if (devptr)hipMemcpy(result.data(), devptr, sizeof(T)*min<const size_t>(length, result.size()), hipMemcpyDeviceToHost);
	return result;
}

template<typename T>Buffer<T>&Buffer<T>::operator=(const std::vector<T>&vector) {
	if (devptr)hipMemcpy(devptr, vector.data(), sizeof(T)*min<const size_t>(length, vector.size()), hipMemcpyHostToDevice);
	return*this;
}

template<typename T>std::size_t Buffer<T>::size()const {
	return length;
}

template<typename T>const T*Buffer<T>::data()const {
	return devptr;
}

template<typename T>void __global__ add(T*const c, const T*a, const T*const b) {
	std::size_t const idx = threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

template<typename T>void __global__ sub(T*const c, const T*a, const T*const b, size_t const length) {
	std::size_t const idx = threadIdx.x;
	if (idx < length)c[idx] = a[idx] - b[idx];
}

template<typename T>void __global__ mul(T*const c, const T*a, const T*const b) {
	std::size_t const idx = threadIdx.x;
	c[idx] = a[idx] * b[idx];
}

template<typename T>void __global__ div(T*const c, const T*a, const T*const b) {
	std::size_t const idx = threadIdx.x;
	c[idx] = a[idx] / b[idx];
}

template<typename T>Buffer<T>operator+(const Buffer<T>&a, const Buffer<T>&b) {
	std::size_t const length = min<std::size_t>(a.size(), b.size());
	Buffer<T>c(length);
	add<T> << <1, length >> > ((T*)c.data(), a.data(), b.data());
	return c;
}

template<typename T>Buffer<T>operator-(const Buffer<T>&a, const Buffer<T>&b) {
	std::size_t const length = min<std::size_t>(a.size(), b.size());
	Buffer<T>c(length);
	sub<T> << <(length - 1) / 64 + 1, 64 >> > ((T*)c.data(), a.data(), b.data(), length);
	return c;
}

template<typename T>Buffer<T>operator*(const Buffer<T>&a, const Buffer<T>&b) {
	std::size_t const length = min<std::size_t>(a.size(), b.size());
	Buffer<T>c(length);
	mul<T> << <1, length >> > ((T*)c.data(), a.data(), b.data());
	return c;
}

template<typename T>Buffer<T>operator/(const Buffer<T>&a, const Buffer<T>&b) {
	std::size_t const length = min<std::size_t>(a.size(), b.size());
	Buffer<T>c(length);
	div<T> << <1, length >> > ((T*)c.data(), a.data(), b.data());
	return c;
}

template<typename T>void __global__ sin(T*const y) {
	std::size_t const idx = threadIdx.x;
	y[idx] = sin(y[idx]);
}

template<typename T>void __global__ cos(T*const y) {
	std::size_t const idx = threadIdx.x;
	y[idx] = cos(y[idx]);
}

template<typename T>void __global__ tan(T*const y) {
	std::size_t const idx = threadIdx.x;
	y[idx] = tan(y[idx]);
}

template<typename T>Buffer<T>sin(const Buffer<T>&x) {
	Buffer<T>y(x);
	sin<T> << <1, y.size() >> > ((T*)y.data());
	return y;
}

template<typename T>Buffer<T>cos(const Buffer<T>&x) {
	Buffer<T>y(x);
	cos<T> << <1, y.size() >> > ((T*)y.data());
	return y;
}

template<typename T>Buffer<T>tan(const Buffer<T>&x) {
	Buffer<T>y(x);
	tan<T> << <1, y.size() >> > ((T*)y.data());
	return y;
}

template<typename T>void __global__ exp(T*const y) {
	std::size_t const idx = threadIdx.x;
	y[idx] = exp(y[idx]);
}

template<typename T>void __global__ log(T*const y) {
	std::size_t const idx = threadIdx.x;
	y[idx] = log(y[idx]);
}

template<typename T>Buffer<T>exp(const Buffer<T>&x) {
	Buffer<T>y(x);
	exp<T> << <1, y.size() >> > ((T*)y.data());
	return y;
}

template<typename T>Buffer<T>log(const Buffer<T>&x) {
	Buffer<T>y(x);
	log<T> << <1, y.size() >> > ((T*)y.data());
	return y;
}


#define T float
template Buffer<T>;
template Buffer<T>operator+(const Buffer<T>&, const Buffer<T>&);
template Buffer<T>operator-(const Buffer<T>&, const Buffer<T>&);
template Buffer<T>operator*(const Buffer<T>&, const Buffer<T>&);
template Buffer<T>operator/(const Buffer<T>&, const Buffer<T>&);
template Buffer<T>sin(const Buffer<T>&);
template Buffer<T>cos(const Buffer<T>&);
template Buffer<T>tan(const Buffer<T>&);
template Buffer<T>exp(const Buffer<T>&);
template Buffer<T>log(const Buffer<T>&);

#undef T

#define T double
template Buffer<T>;
template Buffer<T>operator+(const Buffer<T>&, const Buffer<T>&);
template Buffer<T>operator-(const Buffer<T>&, const Buffer<T>&);
template Buffer<T>operator*(const Buffer<T>&, const Buffer<T>&);
template Buffer<T>operator/(const Buffer<T>&, const Buffer<T>&);
template Buffer<T>sin(const Buffer<T>&);
template Buffer<T>cos(const Buffer<T>&);
template Buffer<T>tan(const Buffer<T>&);
template Buffer<T>exp(const Buffer<T>&);
template Buffer<T>log(const Buffer<T>&);
#undef T
